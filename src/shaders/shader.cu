#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix.h>

#include "math/utils.h"
#include "shared.h"
#include "sampler.h"
#include "path.h"

using namespace krr;	// this is needed or nvcc can't recognize the launchParams external var.

namespace krr
{
	using namespace math;
	using namespace math::utils;
	using namespace shader;
	extern "C" __constant__ LaunchParamsPT launchParams;

	enum {
		SURFACE_RAY_TYPE = 0,
		RAY_TYPE_COUNT
	};

	template <typename... Args>
	KRR_DEVICE_FUNCTION void traceRay(OptixTraversableHandle traversable, Ray ray,
		float tMax, OptixRayFlags flags, Args &&... payload) {

		optixTrace(traversable, ray.origin, ray.dir,
			0.f, tMax, 0.f,						/* ray time val min max */
			OptixVisibilityMask(255),			/* all visible */
			flags,
			SURFACE_RAY_TYPE, RAY_TYPE_COUNT,	/* ray type and number of types */
			SURFACE_RAY_TYPE,					/* miss SBT index */
			std::forward<Args>(payload)...);
	}

	KRR_DEVICE_FUNCTION void handleHit(const ShadingData sd, PathData& path) {
		vec2f r2v = path.sampler.get2D();
		vec3f wiLocal = cosineSampleHemisphere(r2v);
		float bsdfPdf = wiLocal.z * M_1_PI;
		vec3f wi = sd.fromLocal(wiLocal);
		// [NOTE] the generated scattering ray must slightly offseted above the surface to avoid self-intersection
		Ray ray = { sd.pos + sd.N * 1e-3f, wi };
		
		path.ray = ray;
		path.pdf = bsdfPdf;
		//path.throughput *= sd.diffuse * M_1_PI * wiLocal.z / bsdfPdf;
		path.throughput *= sd.diffuse;
		// TODO: direct lighting sampling here
	}

	KRR_DEVICE_FUNCTION void handleMiss() {
		// nothing for now...
	}

	KRR_DEVICE_FUNCTION void prepareShadingData(ShadingData& sd) {
		vec2f barycentric = optixGetTriangleBarycentrics();
		uint primId = optixGetPrimitiveIndex();
		MeshData& mesh = *(MeshData*)optixGetSbtDataPointer();

		sd.wi = -normalize(vec3f(optixGetWorldRayDirection()));
		uint hitKind = optixGetHitKind();
		vec3f bc = { 1 - barycentric.x - barycentric.y, barycentric.x, barycentric.y };
		vec3i triangle = mesh.indices[primId];

		// prepare shading data
		sd.pos = bc.x * mesh.vertices[triangle.x] +
			bc.y * mesh.vertices[triangle.y] +
			bc.z * mesh.vertices[triangle.z];

		sd.geoN = normalize(cross(mesh.vertices[triangle.y] - mesh.vertices[triangle.x],
			mesh.vertices[triangle.z] - mesh.vertices[triangle.x]));

		sd.N = normalize(
			bc.x * mesh.normals[triangle.x] +
			bc.y * mesh.normals[triangle.y] +
			bc.z * mesh.normals[triangle.z]);
		// to do: seems some problem exists with optixIsFrontFaceHit()
		//sd.frontFacing = optixIsFrontFaceHit(hitKind);
		sd.frontFacing = dot(sd.wi, sd.N) > 0.f;
		if (!sd.frontFacing) {
			sd.N = -sd.N;
		}

		if (mesh.tangents != nullptr && mesh.bitangents != nullptr) {
			sd.T = normalize(
				bc.x * mesh.tangents[triangle.x] +
				bc.y * mesh.tangents[triangle.y] +
				bc.z * mesh.tangents[triangle.z]);
			sd.B = normalize(
				bc.x * mesh.bitangents[triangle.x] +
				bc.y * mesh.bitangents[triangle.y] +
				bc.z * mesh.bitangents[triangle.z]);
		}
		else {
			// generate a fake tbn frame for now...
			sd.T = getPerpendicular(sd.N);
			sd.B = normalize(cross(sd.N, sd.T));
		}

		if (mesh.material) {
			Texture& diffuseTexture = mesh.material->mTextures[0];
			hipTextureObject_t cudaTexture = 0;

			if (mesh.texcoords && diffuseTexture.isValid()) {
				cudaTexture = diffuseTexture.getCudaTexture();
				sd.uv = (
					bc.x * mesh.texcoords[triangle.x] +
					bc.y * mesh.texcoords[triangle.y] +
					bc.z * mesh.texcoords[triangle.z]);
				vec4f diffuse = tex2D<float4>(cudaTexture, sd.uv.x, sd.uv.y);
				sd.diffuse = (vec3f)diffuse;
				//sd.diffuse = vec3f(0.7, 0, 0);
			}
			else
				sd.diffuse = vec3f(mesh.material->mMaterialParams.diffuse);
		}
		else {
			sd.diffuse = vec3f(1);
		}
	}

	extern "C" __global__ void KRR_RT_CH(PathTracer)()
	{
		ShadingData& sd = *getPRD<ShadingData>();
		prepareShadingData(sd);

		//sd.emission = 0.2 + 0.8 * dot(sd.N, sd.wi);
		sd.miss = false;
	}

	extern "C" __global__ void KRR_RT_AH(PathTracer)()
	{
		return;
	}

	extern "C" __global__ void KRR_RT_MS(PathTracer)()
	{
		// handle envlighting here, just for now...
		ShadingData &sd = *getPRD<ShadingData>();
		vec3f rayDir = optixGetWorldRayDirection();

		LightSample ls = {};
		ls.wi = rayDir;
		launchParams.envLight.eval(ls);
		sd.emission = ls.Li;
		sd.miss = true;
	}

	KRR_DEVICE_FUNCTION void tracePath(PathData& path) {
		for (uint depth = 0; depth < launchParams.maxDepth; depth++) {
			ShadingData sd = {};
			uint u0, u1;
			packPointer(&sd, u0, u1);
			traceRay(launchParams.traversable, path.ray, 1e20f,
				OPTIX_RAY_FLAG_DISABLE_ANYHIT, u0, u1);
			path.L += path.throughput * sd.emission;

			if (sd.miss) {
				break;
			}
			else {
				handleHit(sd, path);
			}
			// russian roulette
			float u = path.sampler.get1D();
			if (u < launchParams.probRR) break;
			path.throughput /= 1 - launchParams.probRR;
		}

		if (!(path.L < launchParams.clampThreshold))
			path.L = launchParams.clampThreshold;
		// clamp before accumulate?
		path.L = clamp(path.L, vec3f(0), launchParams.clampThreshold);
	}

	extern "C" __global__ void KRR_RT_RG(PathTracer)()
	{
		vec3i launchIndex = optixGetLaunchIndex();
		vec2i pixel = { launchIndex.x, launchIndex.y };

		const int frameID = launchParams.frameID;
		const uint32_t fbIndex = pixel.x + pixel.y * launchParams.fbSize.x;

		Camera& camera = launchParams.camera;
		LCGSampler sampler;
		sampler.setPixel(pixel, frameID);
		// primary ray 
		vec3f rayOrigin = camera.getPosition();
		vec3f rayDir = camera.getRayDir(pixel, launchParams.fbSize);
		PathData path = {};

		vec3f color = vec3f(0);

		for (uint i = 0; i < launchParams.spp; i++) {
			PathData path = {};
			path.sampler = sampler;
			path.ray = { rayOrigin, rayDir };
			tracePath(path);
			color += path.L;
		}

		color /= launchParams.spp;
		launchParams.colorBuffer[fbIndex] = vec4f(color, 1.0f);
	}
}
