#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "LaunchParams.h"
#include "shared.h"

using namespace krr;

namespace krr
{

	// upload updated parameters each rt launch
	extern "C" __constant__ LaunchParams optixLaunchParams;

	enum
	{
		SURFACE_RAY_TYPE = 0,
		RAY_TYPE_COUNT
	};

	extern "C" __global__ void KRR_RT_CH(radiance)()
	{
		vec3f &prd = *getPRD<vec3f>();
		int prim_id = optixGetPrimitiveIndex();
		prd = vec3f(0.5);
	}

	extern "C" __global__ void KRR_RT_AH(radiance)()
	{
	}

	extern "C" __global__ void KRR_RT_MS(radiance)()
	{
		*getPRD<vec3f>() = vec3f(0.1);
	}

	extern "C" __global__ void KRR_RT_RG(renderFrame)()
	{
		vec3i pixelID = optixGetLaunchIndex();
		vec2i pixel = {pixelID.x, pixelID.y};

		const int frameID = optixLaunchParams.frameID;
		const uint32_t fbIndex = pixel.x + pixel.y * optixLaunchParams.fbSize.x;

		Camera &camera = optixLaunchParams.camera;
		vec3f rayDir = camera.getRayDir(pixel, optixLaunchParams.fbSize);

		vec3f prd = vec3f(0);
		uint u0, u1;
		packPointer(&prd, u0, u1);
		optixTrace(optixLaunchParams.traversable,
				   camera.getPosition(),
				   rayDir,
				   0.f,
				   1e10f,
				   0.f,
				   OptixVisibilityMask(255),
				   OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				   SURFACE_RAY_TYPE,
				   RAY_TYPE_COUNT,
				   SURFACE_RAY_TYPE,
				   u0, u1);		// per ray data pointer (32bits each)

		optixLaunchParams.colorBuffer[fbIndex] = vec4f(vec3f(prd), 1.0f);
	}

}
