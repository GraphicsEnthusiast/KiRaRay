#include "hip/hip_runtime.h"
#include "postprocess.h"
#include "math/utils.h"

#include "device/context.h"

KRR_NAMESPACE_BEGIN

using namespace math;
using namespace shader;

namespace shader {

	template<typename T>
    __global__ void accumulateFrame(uint n_elements, vec4f* currentBuffer, vec4f* accumBuffer, uint accumCount, bool average)
    {
		uint i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= n_elements) return;
		float currentWeight = 1.f / (accumCount + 1);
		if (accumCount > 0) {
			if(average)
				// moving average mode
				accumBuffer[i] = utils::lerp(accumBuffer[i], currentBuffer[i], currentWeight);
			else
				// sum mode
				accumBuffer[i] = accumBuffer[i] + currentBuffer[i];
		}
		else {
			accumBuffer[i] = currentBuffer[i];
		}
		if (average)
			currentBuffer[i] = accumBuffer[i];
		else
			currentBuffer[i] = accumBuffer[i] * currentWeight;
	}
}

void AccumulatePass::render(CUDABuffer& frame) {

	if (!mEnable) return;
	if (mpScene->getChanges()) reset();
	hipStream_t& stream = gpContext->cudaStream;
	linear_kernel(accumulateFrame<vec4f>, 0, stream, mFrameSize.x * mFrameSize.y, 
		(vec4f*)frame.data(), (vec4f*)mAccumBuffer.data(), mAccumCount, false);

	mAccumCount = min(mAccumCount + 1, mMaxAccumCount - 1);
}

namespace shader {
	namespace tonemapper {
		KRR_CALLABLE vec3f toneMapAces(vec3f color) {
			// https://knarkowicz.wordpress.com/2016/01/06/aces-filmic-tone-mapping-curve/
			color *= 0.6;
			float A = 2.51;
			float B = 0.03;
			float C = 2.43;
			float D = 0.59;
			float E = 0.14;
			color = saturate((color * (A * color + B)) / (color * (C * color + D) + E));
			return color;
		}

		KRR_CALLABLE vec3f toneMapReinhard(vec3f color)
		{
			float luminance = utils::luminance(color);
			float reinhard = luminance / (luminance + 1);
			return color * (reinhard / luminance);
		}

		KRR_CALLABLE vec3f toneMapUC2(vec3f color) {
			float A = 0.22; // Shoulder Strength
			float B = 0.3;  // Linear Strength
			float C = 0.1;  // Linear Angle
			float D = 0.2;  // Toe Strength
			float E = 0.01; // Toe Numerator
			float F = 0.3;  // Toe Denominator

			color = ((color * (A * color + C * B) + D * E) / (color * (A * color + B) + D * F)) - (E / F);
			return color;
		}

		KRR_CALLABLE vec3f toneMapHejiHableAlu(vec3f color)
		{
			color = max(vec3f(0), color - 0.004f);
			color = (color * (6.2f * color + 0.5f)) / (color * (6.2f * color + 1.7f) + 0.06f);
			// Result includes sRGB conversion
			return pow(color, vec3f(2.2));
		}
	}

	using namespace tonemapper;

	template <typename T>
	__global__ void toneMap(uint n_elements, vec4f* frame, vec3f colorTransform, ToneMappingPass::Operator toneMapOperator) {
		uint i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= n_elements) return;
		vec3f color = vec3f(frame[i]) * colorTransform;
		switch (toneMapOperator)
		{
		case krr::ToneMappingPass::Operator::Linear:
			break;
		case krr::ToneMappingPass::Operator::Reinhard:
			color = toneMapReinhard(color);
			break;
		case krr::ToneMappingPass::Operator::Aces:
			color = toneMapAces(color);
			break;
		case krr::ToneMappingPass::Operator::Uncharted2:
			color = toneMapUC2(color);
			break;
		case krr::ToneMappingPass::Operator::HejiHable:
			color = toneMapHejiHableAlu(color);
			break;
		default:
			break;
		}
		frame[i] = vec4f(color, 1.f);
	}
}

void ToneMappingPass::render(CUDABuffer& frame)
{
	if (!mEnable) return;
	hipStream_t &stream = gpContext->cudaStream;
	vec3f colorTransform = vec3f(mExposureCompensation);
	linear_kernel(toneMap<float>, 0, stream, mFrameSize.x * mFrameSize.y,
		(vec4f*)frame.data(), colorTransform, mOperator);
	
}

KRR_NAMESPACE_END


